#include "hip/hip_runtime.h"
/*
rgbd-tracker
Copyright (c) 2014, Tommi Tykkälä, All rights reserved.

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 3.0 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library.
*/
#include <image2/Image2.h>
#include <image2/ImagePyramid2.h>
#include "hostUtils.h"
#include <stdio.h>
#include <cwchar>

#include <calib/calib.h>
namespace d2zutils {
	#include "kernelUtils.h"
}
#include <rendering/VertexBuffer2.h>

using namespace d2zutils;

__global__ void d2ZKernel( unsigned short *dPtr, float *zPtr,int width, int height, float *calibDataDev/*, float *vData, unsigned char *imgData*/)
{
	int xi = blockIdx.x*blockDim.x+threadIdx.x;
	int yi = blockIdx.y*blockDim.y+threadIdx.y;
	int dstIdx = xi+yi*width;
	// IR image -> disparity image has constant offset (Konolige's tech guide)
	// http://www.ros.org/wiki/kinect_calibration/technical
	unsigned int sxi = xi - 4;
	unsigned int syi = yi - 3;
	if (sxi < width && syi < height) {
		int srcIdx = sxi + syi * width;
            //    float fx = calibDataDev[KL_OFFSET];
        float c0 = calibDataDev[C0_OFFSET];
        float c1 = calibDataDev[C1_OFFSET];

      //          float b = calibDataDev[b_OFFSET];
//                float B = calibDataDev[B_OFFSET];
		//float minDist = calibDataDev[9*2+5+16+1+2*9+1];
                float maxDist = calibDataDev[MAXD_OFFSET];
		float d = (float)dPtr[srcIdx];
//		float *T = &calibDataDev[9*2+5];
//		float *KR = &calibDataDev[0];
		if (d > 0 && d < 2047) {
            float z = fabs(1.0f/(c0+c1*d));
            //float z = fabs(8.0f*b*fx/(B-d));
			if (z > maxDist) z = 0.0f;
			zPtr[dstIdx] = z/maxDist;		
		/*	float3 p3,r3,p2; 
			p3.x = -(float(xi) - cx) * z / fx;
			p3.y = -(float(yi) - cy) * z / fy;
			p3.z = -z;
			matrixMultVec4(T, p3, r3);
			vData[dstIdx*6+0] = r3.x; 
			vData[dstIdx*6+1] = r3.y;
			vData[dstIdx*6+2] = r3.z;

			matrixMultVec3(KR, r3, p2); p2.x /= p2.z; p2.y /= p2.z;

			unsigned char color = 0;
			bilinearInterpolation(p2, width, height, imgData, color);
			float colorF = float(color)/255.0f;
			vData[dstIdx*6+3] = colorF;
			vData[dstIdx*6+4] = colorF;
			vData[dstIdx*6+5] = colorF;*/
			return;
		} 
	}
	zPtr[dstIdx] = 0.0f;
/*	vData[dstIdx*6+0] = 0;
	vData[dstIdx*6+1] = 0;
	vData[dstIdx*6+2] = 0;
	vData[dstIdx*6+3] = 0.0f;
	vData[dstIdx*6+4] = 0.0f;
	vData[dstIdx*6+5] = 0.0f;*/
}


__global__ void undistortDisparityKernel( unsigned short *dPtr, float *uPtr,int width, int height, float *calibDataDev)
{
    int xi = blockIdx.x*blockDim.x+threadIdx.x;
    int yi = blockIdx.y*blockDim.y+threadIdx.y;
    int idx = xi+yi*width;

    float alpha0 = calibDataDev[ALPHA0_OFFSET];
    float alpha1 = calibDataDev[ALPHA1_OFFSET];
    float *beta  = &calibDataDev[BETA_OFFSET];

    float d = (float)dPtr[idx];
    float ud = 0xffff;
    if (d < 2047) {
        ud = d + beta[idx]*expf(alpha0-alpha1*d);
    }
    uPtr[idx] = ud;
}



__global__ void d2ZLowKernel( unsigned short *dPtr, float *zPtr, int width, int height, float *calibDataDev, float xOff, float yOff)
{
    int xi = blockIdx.x*blockDim.x+threadIdx.x;
    int yi = blockIdx.y*blockDim.y+threadIdx.y;
    int dstIdx = xi+yi*(width/2);

    // IR image -> disparity image has constant offset (Konolige's tech guide)
    // http://www.ros.org/wiki/kinect_calibration/technical
    unsigned int sxi = 2*xi + xOff;
    unsigned int syi = 2*yi + yOff;
    if (sxi < width-1 && syi < height-1) {
        int srcIdx1 = sxi + 0 + (syi + 0) * width;
        int srcIdx2 = sxi + 1 + (syi + 0) * width;
        int srcIdx3 = sxi + 1 + (syi + 1) * width;
        int srcIdx4 = sxi + 0 + (syi + 1) * width;
        // fx and B manually *2 because they RGB_WIDTH_SMALL*2 = DISPARITY_WIDTH (set in the main program according to rgb)
        float c0 = calibDataDev[C0_OFFSET];
        float c1 = calibDataDev[C1_OFFSET];
        //float fx = calibDataDev[KL_OFFSET]*2;
        //float B = calibDataDev[B_OFFSET]*2;
        // the rest values read normally as they are reso invariant
        //float b = calibDataDev[b_OFFSET];
        float minDist = calibDataDev[MIND_OFFSET];
        float maxDist = calibDataDev[MAXD_OFFSET];

        float d1 = (float)dPtr[srcIdx1];
        float d2 = (float)dPtr[srcIdx2];
        float d3 = (float)dPtr[srcIdx3];
        float d4 = (float)dPtr[srcIdx4];
        if ((d1 < 2047) && (d2 < 2047) && (d3 < 2047) && (d4 < 2047)) {
            //			float d = (d1+d2+d3+d4)/4.0f;
            float d = d1;
            if (d2 < d) d = d2;
            if (d3 < d) d = d3;
            if (d4 < d) d = d4;
//            float z = fabs(8.0f*b*fx/(B-d));
            float z = fabs(1.0f/(c0+c1*d));
            if (z > maxDist || z < minDist) z = 0.0f;
            zPtr[dstIdx] = (z-minDist)/(maxDist-minDist);
            return;
        }
    }
    zPtr[dstIdx] = 0.0f;
}


__global__ void d2ZLowHdrKernel( float *dPtr, float *zPtr, int width, int height, float *calibDataDev, float xOff, float yOff)
{
    int xi = blockIdx.x*blockDim.x+threadIdx.x;
    int yi = blockIdx.y*blockDim.y+threadIdx.y;
    int dstIdx = xi+yi*(width/2);

    // IR image -> disparity image has constant offset (Konolige's tech guide)
    // http://www.ros.org/wiki/kinect_calibration/technical
    unsigned int sxi = 2*xi + xOff;
    unsigned int syi = 2*yi + yOff;
    if (sxi < width-1 && syi < height-1) {
        int srcIdx1 = sxi + 0 + (syi + 0) * width;
        int srcIdx2 = sxi + 1 + (syi + 0) * width;
        int srcIdx3 = sxi + 1 + (syi + 1) * width;
        int srcIdx4 = sxi + 0 + (syi + 1) * width;
        // fx and B manually *2 because they RGB_WIDTH_SMALL*2 = DISPARITY_WIDTH (set in the main program according to rgb)
        float c0 = calibDataDev[C0_OFFSET];
        float c1 = calibDataDev[C1_OFFSET];
        //float fx = calibDataDev[KL_OFFSET]*2;
        //float B = calibDataDev[B_OFFSET]*2;
        // the rest values read normally as they are reso invariant
        //float b = calibDataDev[b_OFFSET];
        float minDist = calibDataDev[MIND_OFFSET];
        float maxDist = calibDataDev[MAXD_OFFSET];

        float d1 = dPtr[srcIdx1];
        float d2 = dPtr[srcIdx2];
        float d3 = dPtr[srcIdx3];
        float d4 = dPtr[srcIdx4];
        if ((d1 < 2047) && (d2 < 2047) && (d3 < 2047) && (d4 < 2047)) {
            //			float d = (d1+d2+d3+d4)/4.0f;
            float d = d1;
            if (d2 < d) d = d2;
            if (d3 < d) d = d3;
            if (d4 < d) d = d4;
            float z = fabs(1.0f/(c0+c1*d));
            if (z > maxDist || z < minDist) z = 0.0f;
            zPtr[dstIdx] = (z-minDist)/(maxDist-minDist);
            return;
        }
    }
    zPtr[dstIdx] = 0.0f;
}

__global__ void setMaxZKernel(float *zPtr, float *calibDataDev) {
    int offset = blockIdx.x*blockDim.x+threadIdx.x;
    float maxDist = calibDataDev[MAXD_OFFSET];
    zPtr[offset] = maxDist;
}



__global__ void z2CloudKernel(float *zPtr,int width, int height, float *calibDataDev, float *vData, float *rgbData, float *imgData1, float *zPtrDst, int stride)
{
    int xi = blockIdx.x*blockDim.x+threadIdx.x;
    int yi = blockIdx.y*blockDim.y+threadIdx.y;
    int offset = xi+yi*width;
    // make sure stride has matching number of elements stored here!
    int idxStride = offset*stride;
    float z   = zPtr[offset];
    float maxDist = calibDataDev[MAXD_OFFSET];
    if (z > 0) {
        float fx      = calibDataDev[KL_OFFSET];
        float fy      = calibDataDev[KL_OFFSET+4];
        float cx      = calibDataDev[KL_OFFSET+2];
        float cy      = calibDataDev[KL_OFFSET+5];
        float minDist = calibDataDev[MIND_OFFSET];
        float *T      = &calibDataDev[TLR_OFFSET];
        float *KR     = &calibDataDev[KR_OFFSET];
        float *kc     = &calibDataDev[KcR_OFFSET];

        z   = -(z*(maxDist-minDist) + minDist);

        float3 p3,r3;
        p3.x = (float(xi) - cx) * z / fx;
        p3.y = (float(yi) - cy) * z / fy;
        p3.z = z;
        matrixMultVec4(T, p3, r3);

        float2 pu,p2_1;
        pu.x = r3.x / r3.z;
        pu.y = r3.y / r3.z;

        distortPoint(pu,kc,KR,p2_1);

        bool pointsOnScreen = true;
        if (!inBounds(p2_1,width,height)) pointsOnScreen = false;

        float colorR1 = 0, colorG1 = 0, colorB1 = 0;
        float color1  = 0;
        float gradX1  = 0, gradY1  = 0;
        if (pointsOnScreen) {
            int xdi,ydi;
            float fx,fy;
            xdi = (int)p2_1.x; ydi = (int)p2_1.y; fx = p2_1.x - xdi; fy = p2_1.y - ydi;
            // interpolate rgb color
            bilinearInterpolation(xdi,   ydi,   fx, fy, width, rgbData, colorR1,colorG1,colorB1);
            // faster to compute gray value from RGB than bilinear interpolation:
            color1 = 0.3f*colorR1 + 0.59f*colorG1 + 0.11f*colorB1;

            int zoff = xdi+ydi*width;

            float nZ = (-r3.z-minDist)/(maxDist-minDist);
            zPtrDst[zoff] = nZ;
            zPtrDst[zoff+1] = nZ;
            zPtrDst[zoff+width] = nZ;
            zPtrDst[zoff+width+1] = nZ;

            // interpolate gradient
            float colorN = 0, colorS = 0, colorE = 0, colorW = 0;

            xdi = (int)p2_1.x; ydi = (int)(p2_1.y-1.0f);
            bilinearInterpolation(xdi, ydi, fx, fy, width, imgData1, colorN);

            xdi = (int)(p2_1.x+1.0f); ydi = (int)p2_1.y;
            bilinearInterpolation(xdi, ydi,   fx, fy, width, imgData1, colorE);

            xdi = (int)(p2_1.x-1.0f); ydi = (int)p2_1.y;
            bilinearInterpolation(xdi, ydi,   fx, fy, width, imgData1, colorW);

            xdi = (int)p2_1.x; ydi = (int)(p2_1.y+1.0f);
            bilinearInterpolation(xdi,   ydi, fx, fy, width, imgData1, colorS);
            gradX1 = (colorE-colorW)/2.0f;
            gradY1 = (colorS-colorN)/2.0f;

            vData[idxStride+0]  = r3.x;
            vData[idxStride+1]  = r3.y;
            vData[idxStride+2]  = r3.z;
            vData[idxStride+3]  = 0.0f;//n.x
            vData[idxStride+4]  = 0.0f;//n.y
            vData[idxStride+5]  = 0.0f;//n.z
            vData[idxStride+6]  = p2_1.x;
            vData[idxStride+7]  = p2_1.y;
            vData[idxStride+8]  = colorR1;
            vData[idxStride+9]  = colorG1;
            vData[idxStride+10] = colorB1;
            vData[idxStride+11] = gradX1; // store gradientX for reference image based optimization
            vData[idxStride+12] = gradY1; // store gradientY for reference image based optimization
            vData[idxStride+13] = min(fabs(gradY1)+fabs(gradX1),1.0f); // store gradient magnitude for thresholding, range: [0,1]
            vData[idxStride+14] = color1;
            //            vData[idxStride+15] = gradX2;
            //            vData[idxStride+16] = gradY2;
            //            vData[idxStride+17] = color2;
            //            vData[idxStride+18] = gradX3;
            //            vData[idxStride+19] = gradY3;
            //            vData[idxStride+20] = color3;
            return;
        }
    }

    vData[idxStride+0]  = 0.0f;
    vData[idxStride+1]  = 0.0f;
    vData[idxStride+2]  = -maxDist; // set depth to a large value for ensuring big depth discrepancy for zweighting
    vData[idxStride+3]  = 0.0f;
    vData[idxStride+4]  = 0.0f;
    vData[idxStride+5]  = 0.0f;
    vData[idxStride+6]  = 0.0f;
    vData[idxStride+7]  = 0.0f;
    vData[idxStride+8]  = 0.0f;
    vData[idxStride+9]  = 0.0f;
    vData[idxStride+10] = 0.0f;
    vData[idxStride+11] = 0.0f;
    vData[idxStride+12] = 0.0f;
    vData[idxStride+13] = 0.0f;
    vData[idxStride+14] = 0.0f;
    vData[idxStride+15] = 0.0f;
    vData[idxStride+16] = 0.0f;
    vData[idxStride+17] = 0.0f;
    vData[idxStride+18] = 0.0f;
    vData[idxStride+19] = 0.0f;
    vData[idxStride+20] = 0.0f;

}

__global__ void z2CloudKernelFast(float *zPtr,int width, int height, float *calibDataDev, float *vData, float *rgbData, float *imgData1, float *zPtrDst, int stride)
{
    int xi = blockIdx.x*blockDim.x+threadIdx.x;
    int yi = blockIdx.y*blockDim.y+threadIdx.y;
    int offset = xi+yi*width;
    // make sure stride has matching number of elements stored here!
    int idxStride = offset*stride;
    float z   = zPtr[offset];
    float maxDist = calibDataDev[MAXD_OFFSET];
    if (z > 0) {
            float fx      = calibDataDev[KL_OFFSET];
            float fy      = calibDataDev[KL_OFFSET+4];
            float cx      = calibDataDev[KL_OFFSET+2];
            float cy      = calibDataDev[KL_OFFSET+5];
            float minDist = calibDataDev[MIND_OFFSET];
            float *T      = &calibDataDev[TLR_OFFSET];
            float *KR     = &calibDataDev[KR_OFFSET];
            float *kc     = &calibDataDev[KcR_OFFSET];

            z   = -(z*(maxDist-minDist) + minDist);

            float3 p3,r3;
            p3.x = (float(xi) - cx) * z / fx;
            p3.y = (float(yi) - cy) * z / fy;
            p3.z = z;
            matrixMultVec4(T, p3, r3);

            float2 pu,p2_1;
            pu.x = r3.x / r3.z;
            pu.y = r3.y / r3.z;
            distortPoint(pu,kc,KR,p2_1);

            float colorR1 = 0, colorG1 = 0, colorB1 = 0;
            float color1  = 0;

            int xdi = (int)p2_1.x;
            int ydi = (int)p2_1.y;

            if (xdi >= 0 && ydi >= 0 && xdi <= width-2 && ydi <= height-2)
            {
                // xdi in [2,width - 4]
                // ydi in [2,height- 4]
                // -> (p2.x,p2.y) maps into valid lowres domain too + bilinear interpolation
                float fx = p2_1.x - xdi;
                float fy = p2_1.y - ydi;
                // interpolate rgb color
                bilinearInterpolation(xdi,   ydi,   fx, fy, width, rgbData, colorR1,colorG1,colorB1);
                // faster to compute gray value from RGB than bilinear interpolation:
                color1 = 0.3f*colorR1 + 0.59f*colorG1 + 0.11f*colorB1;
                int zoff = xdi+ydi*width;

                float nZ = (-r3.z-minDist)/(maxDist-minDist);
                zPtrDst[zoff] = nZ;
                zPtrDst[zoff+1] = nZ;
                zPtrDst[zoff+width] = nZ;
                zPtrDst[zoff+width+1] = nZ;
                vData[idxStride+0]  = r3.x;
                vData[idxStride+1]  = r3.y;
                vData[idxStride+2]  = r3.z;
                vData[idxStride+3]  = 0.0f; //n.x
                vData[idxStride+4]  = 0.0f; //n.y
                vData[idxStride+5]  = 0.0f; //n.z
                vData[idxStride+6]  = p2_1.x;
                vData[idxStride+7]  = p2_1.y;
                vData[idxStride+8]  = colorR1;//n.x;
                vData[idxStride+9]  = colorG1;//n.y;
                vData[idxStride+10] = colorB1;//n.z;
                vData[idxStride+11] = 0;
                vData[idxStride+12] = 0;
                vData[idxStride+13] = 0.0f; // store zero gradient magnitude (no pixel selection support in this method)
                vData[idxStride+14] = color1;
    //            vData[idxStride+15] = gradX2;
    //            vData[idxStride+16] = gradY2;
    //            vData[idxStride+17] = color2;
    //            vData[idxStride+18] = gradX3;
    //            vData[idxStride+19] = gradY3;
    //            vData[idxStride+20] = color3;
                return;
            }
    }

    vData[idxStride+0]  = 0.0f;
    vData[idxStride+1]  = 0.0f;
    vData[idxStride+2]  = -maxDist; // set depth to a large value for ensuring big depth discrepancy for zweighting
    vData[idxStride+3]  = 0.0f;
    vData[idxStride+4]  = 0.0f;
    vData[idxStride+5]  = 0.0f;
    vData[idxStride+6]  = 0.0f;
    vData[idxStride+7]  = 0.0f;
    vData[idxStride+8]  = 0.0f;
    vData[idxStride+9]  = 0.0f;
    vData[idxStride+10] = 0.0f;
    vData[idxStride+11] = 0.0f;
    vData[idxStride+12] = 0.0f;
    vData[idxStride+13] = 0.0f;
    vData[idxStride+14] = 0.0f;
    vData[idxStride+15] = 0.0f;
    vData[idxStride+16] = 0.0f;
    vData[idxStride+17] = 0.0f;
    vData[idxStride+18] = 0.0f;
    vData[idxStride+19] = 0.0f;
    vData[idxStride+20] = 0.0f;

}

__global__ void setNormalsCudaKernel(float *vertexData,float *normalData, float scale, int stride) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idxStride = idx*stride;
    float x = vertexData[idxStride+0];
    float y = vertexData[idxStride+1];
    float z = vertexData[idxStride+2];
    vertexData[idxStride+3] = x+normalData[idx*3+0]*scale;
    vertexData[idxStride+4] = y+normalData[idx*3+1]*scale;
    vertexData[idxStride+5] = z+normalData[idx*3+2]*scale;
}


__global__ void extractGradientKernel(float *vertexData, int stride, int slot, float *gradientScratchDev) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idxStride = idx*stride;
    float gradMag = vertexData[idxStride+slot];
    gradientScratchDev[idx] = gradMag;
}

__global__ void addVertexAttributesKernel(int *indexPointer, float *vData, float *zPtr, int width, int height, float *calibDataDev, float *imgData1, float *imgData2, float *imgData3, int stride)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int offset = indexPointer[idx];
    // make sure stride has matching number of elements stored here!
    int idxStride = offset*stride;
    // zvalues for computing point normal

    float *kc     = &calibDataDev[KcR_OFFSET];
    float *KR     = &calibDataDev[KR_OFFSET];

    float3 r3;
    r3.x = vData[idxStride+0];
    r3.y = vData[idxStride+1];
    r3.z = vData[idxStride+2];

    /*
    float zNu = zPtr[offset+1];
    float zNv = zPtr[offset+width];
    zNu   = -(zNu*(maxDist-minDist) + minDist);
    zNv   = -(zNv*(maxDist-minDist) + minDist);


    float3 p3;
    p3.x = (float(xi+1) - cx) * zNu / fx;
    p3.y = (float(yi+0) - cy) * zNu / fy;
    p3.z = zNu;
    matrixMultVec4(T, p3, r3u);

    p3.x = (float(xi+0) - cx) * zNv / fx;
    p3.y = (float(yi+1) - cy) * zNv / fy;
    p3.z = zNv;
    matrixMultVec4(T, p3, r3v);

    float3 nu,nv,n;
    nu.x = r3u.x - r3.x; nu.y = r3u.y - r3.y; nu.z = r3u.z - r3.z;
    nv.x = r3v.x - r3.x; nv.y = r3v.y - r3.y; nv.z = r3v.z - r3.z;
    // compute normal as crossproduct
    n.x =  nu.y * nv.z - nu.z * nv.y;
    n.y =-(nu.x * nv.z - nu.z * nv.x);
    n.z =  nu.x * nv.y - nu.y * nv.x;
    // normal to unit length
    float len = sqrt(n.x*n.x + n.y*n.y + n.z*n.z+1e-5f);
    // TODO: use this magnitude (area of square) to prune out invalid normals (mismatch in depth)
    n.x /= len; n.y /= len; n.z /= len;
    */
    float color2  = 0, color3  = 0;
    float colorN,colorW,colorE,colorS;
    float gradX2  = 0, gradY2  = 0;
    float gradX3  = 0, gradY3  = 0;

    float2 p_1,p_2,p_3;

    float2 pu;
    pu.x = r3.x / r3.z;
    pu.y = r3.y / r3.z;
    distortPoint(pu,kc,KR,p_1);

    bool pointsOnScreen = true;
    if (!inBounds(p_1,width,height)) pointsOnScreen = false;

    if (pointsOnScreen) {
       // compute low-resolution coordinates
        float a = 0.5f; float b = -0.25f;
        p_2.x  = a*p_1.x  + b; p_2.y = a*p_1.y + b;

        p_3.x  = a*p_2.x  + b; p_3.y  = a*p_2.y + b;

        int xdi,ydi;
        float fracX,fracY;

        xdi = (int)p_2.x; fracX = p_2.x - xdi;
        ydi = (int)p_2.y; fracY = p_2.y - ydi;
        bilinearInterpolation(xdi,   ydi,   fracX, fracY, width/2, imgData2, color2);

        xdi = (int)p_2.x; 
        ydi = (int)p_2.y-1; 
        bilinearInterpolation(xdi, ydi, fracX, fracY, width/2, imgData2, colorN);

        xdi = (int)p_2.x-1; 
        ydi = (int)p_2.y; 
        bilinearInterpolation(xdi, ydi, fracX, fracY, width/2, imgData2, colorW);

        xdi = (int)p_2.x+1; 
        ydi = (int)p_2.y; 
        bilinearInterpolation(xdi, ydi, fracX, fracY, width/2, imgData2, colorE);

        xdi = (int)p_2.x; 
        ydi = (int)p_2.y+1; 
        bilinearInterpolation(xdi, ydi, fracX, fracY, width/2, imgData2, colorS);

        gradX2 = (colorE-colorW)/2.0f;
        gradY2 = (colorS-colorN)/2.0f;

        xdi = (int)p_3.x; fracX = p_3.x - xdi;
        ydi = (int)p_3.y; fracY = p_3.y - ydi;
        bilinearInterpolation(xdi,   ydi,   fracX, fracY, width/4, imgData3, color3);

        xdi = (int)p_3.x; 
        ydi = (int)p_3.y-1; 
        bilinearInterpolation(xdi,   ydi, fracX, fracY, width/4, imgData3, colorN);

        xdi = (int)p_3.x-1; 
        ydi = (int)p_3.y; 
        bilinearInterpolation(xdi, ydi,   fracX, fracY, width/4, imgData3, colorW);

        xdi = (int)p_3.x+1; 
        ydi = (int)p_3.y; 
        bilinearInterpolation(xdi, ydi,   fracX, fracY, width/4, imgData3, colorE);

        xdi = (int)p_3.x; 
        ydi = (int)p_3.y+1; 
        bilinearInterpolation(xdi,   ydi, fracX, fracY, width/4, imgData3, colorS);

       gradX3 = (colorE-colorW)/2.0f;
       gradY3 = (colorS-colorN)/2.0f;
    }
    // normal points are currently computed on CPU only for keyframes
    //vData[idxStride+3]  = r3.x - n.x*100.0f;
    //vData[idxStride+4]  = r3.y - n.y*100.0f;
    //vData[idxStride+5]  = r3.z - n.z*100.0f;
//  vData[idxStride+11] = gradX1; // store gradientX for reference image based optimization
//  vData[idxStride+12] = gradY1; // store gradientY for reference image based optimization
//  vData[idxStride+13] = 127.9f*(fabs(gradY1)+fabs(gradX1)); // store gradient magnitude for thresholding, int range: [0,255]
//  vData[idxStride+14] = color1;

    vData[idxStride+15] = gradX2;
    vData[idxStride+16] = gradY2;
    vData[idxStride+17] = color2;
    vData[idxStride+18] = gradX3;
    vData[idxStride+19] = gradY3;
    vData[idxStride+20] = color3;
}

extern "C" void d2ZCuda(unsigned short *disparity16U, Image2 *zImage, float *calibDataDev, float xOff, float yOff)
{
    if (disparity16U == 0 || zImage == 0 || zImage->devPtr == NULL || calibDataDev == NULL) return;
	float *zPtr= (float*)zImage->devPtr;
	dim3 cudaBlockSize(32,30,1);
    dim3 cudaGridSize(zImage->width/cudaBlockSize.x,zImage->height/cudaBlockSize.y,1);
    d2ZLowKernel<<<cudaGridSize,cudaBlockSize,0,zImage->cudaStream>>>(disparity16U,zPtr,zImage->width*2,zImage->height*2,calibDataDev, xOff, yOff);
}

extern "C" void d2ZCudaHdr(float *disparityHdr, Image2 *zImage, float *calibDataDev, float xOff, float yOff) {
    if (disparityHdr == 0 || zImage == 0 || zImage->devPtr == NULL || calibDataDev == NULL) return;
    float *zPtr= (float*)zImage->devPtr;
    dim3 cudaBlockSize(32,30,1);
    dim3 cudaGridSize(zImage->width/cudaBlockSize.x,zImage->height/cudaBlockSize.y,1);
    d2ZLowHdrKernel<<<cudaGridSize,cudaBlockSize,0,zImage->cudaStream>>>(disparityHdr,zPtr,zImage->width*2,zImage->height*2,calibDataDev, xOff,yOff);
}

extern "C" void undistortDisparityCuda(unsigned short *disparity16U, float *uPtr, float *calibDataDev, int width, int height, hipStream_t stream = 0)
{
    if (disparity16U == 0 || uPtr == NULL || calibDataDev == NULL) return;
    dim3 cudaBlockSize(32,30,1);
    dim3 cudaGridSize(width/cudaBlockSize.x,height/cudaBlockSize.y,1);
    undistortDisparityKernel<<<cudaGridSize,cudaBlockSize,0,stream>>>(disparity16U,uPtr,width,height,calibDataDev);
}


extern "C" void z2CloudCuda(Image2 *zImageIR, float *calibDataDev, VertexBuffer2 *vbuffer, Image2 *rgbImage, ImagePyramid2 *grayPyramid, Image2 *zImage, bool computeGradients)
{
    if (zImageIR == 0 || zImageIR->devPtr == NULL || zImage == 0 || zImage->devPtr == NULL || calibDataDev == NULL || vbuffer == NULL || vbuffer->devPtr == NULL || rgbImage == NULL || rgbImage->devPtr == NULL || grayPyramid == NULL) {
        printf("null given to z2CloudCuda!\n");
        if (zImage->devPtr == NULL) printf("zImage not locked!\n");
        fflush(stdin);
        fflush(stdout);
        return;
    }
    float *imgData = (float*)grayPyramid->getImageRef(0).devPtr;
    if (imgData == NULL) {
        return;
    }
    float *zPtr= (float*)zImageIR->devPtr;
    float *zPtrDst= (float*)zImage->devPtr;
    float *vData = (float*)vbuffer->devPtr;
    float *rgbData = (float*)rgbImage->devPtr;
    dim3 cudaBlockSize(32,15,1);
    dim3 cudaGridSize(zImage->width/cudaBlockSize.x,zImage->height/cudaBlockSize.y,1);
    vbuffer->setVertexAmount(zImage->width * zImage->height);
    if (computeGradients) {
        z2CloudKernel<<<cudaGridSize,cudaBlockSize,0,vbuffer->cudaStream>>>(zPtr,zImageIR->width,zImageIR->height,calibDataDev,vData,rgbData,imgData,zPtrDst,vbuffer->getStride());
    } else {
        z2CloudKernelFast<<<cudaGridSize,cudaBlockSize,0,vbuffer->cudaStream>>>(zPtr,zImageIR->width,zImageIR->height,calibDataDev,vData,rgbData,imgData,zPtrDst,vbuffer->getStride());
    }
    checkCudaError("z2CloudCuda error");
}

extern "C" void addVertexAttributesCuda(Image2 *zImage, float *calibDataDev, VertexBuffer2 *vbuffer, ImagePyramid2 *grayPyramid)
{
    if (vbuffer == NULL || vbuffer->devPtr == NULL || vbuffer->indexDevPtr == NULL || grayPyramid == NULL || calibDataDev == NULL || zImage == NULL || zImage->devPtr == NULL) {
        printf("addVertexAttributesCuda: null pointer given!\n"); return;
    }

    float *imgData[3];
    assert(grayPyramid->nLayers == 3);
    for (int i = 0; i < 3; i++) {
        imgData[i] = (float*)grayPyramid->getImageRef(i).devPtr;
        if (imgData[i] == NULL) {
            printf("addVertexAttributesCuda error: grayPyramid layer %d not locked! panik exit \n",i);
            return;
        }
    }

     // enforce multiple of 1024 for element count -> max performance
     if (vbuffer->getElementsCount()%512 != 0) {
          printf("addVertexAttributesCuda: vbuffer has wrong number of selected pixels! (%d)\n",vbuffer->getElementsCount());
          return;
    }

    float *zPtr= (float*)zImage->devPtr;
    int *indexPointer = (int*)vbuffer->indexDevPtr;
    float *vertexData = (float*)vbuffer->devPtr;
    int nElements = vbuffer->getElementsCount();
    dim3 cudaBlockSize(512,1,1);
    dim3 cudaGridSize(nElements/cudaBlockSize.x,1,1);
    addVertexAttributesKernel<<<cudaGridSize,cudaBlockSize,0,vbuffer->cudaStream>>>(indexPointer,vertexData,zPtr,zImage->width,zImage->height,calibDataDev,imgData[0],imgData[1],imgData[2],vbuffer->getStride());
    checkCudaError("addVertexAttributesCuda error");
}

extern "C" void setNormalsCuda(VertexBuffer2 *vbuffer, float *normalData, float scale) {
    if (vbuffer == NULL || vbuffer->devPtr == NULL || normalData == NULL) {
        printf("setNormalsCuda: null pointer given!\n"); return;
    }

     // enforce multiple of 1024 for element count -> max performance
     if (vbuffer->getVertexCount()%1024 != 0) {
          printf("setNormalsCuda: vbuffer has wrong number of vertices! (%d)\n",vbuffer->getVertexCount());
          return;
    }

    float *vertexData = (float*)vbuffer->devPtr;
    int nElements = vbuffer->getVertexCount();
    dim3 cudaBlockSize(1024,1,1);
    dim3 cudaGridSize(nElements/cudaBlockSize.x,1,1);
    setNormalsCudaKernel<<<cudaGridSize,cudaBlockSize,0,vbuffer->cudaStream>>>(vertexData,normalData,scale,vbuffer->getStride());
    checkCudaError("setNormalsCuda error");
}

extern "C" void extractGradientMagnitudes(VertexBuffer2 *vbuffer, float *gradientScratchDev)
{
    if (vbuffer == NULL || vbuffer->devPtr == NULL || gradientScratchDev == NULL) {
        printf("extractGradientMagnitudes: null pointer given!\n"); return;
    }

     // enforce multiple of 1024 for element count -> max performance
     if (vbuffer->getVertexCount()%1024 != 0) {
          printf("extractGradientMagnitudes: vbuffer has wrong number of vertices! (%d)\n",vbuffer->getVertexCount());
          return;
    }

     if (vbuffer->getStride() != VERTEXBUFFER_STRIDE) {
         printf("extractGradientMagnitudes: vertexbuffer has illegal stride (%d), must be %d!\n",vbuffer->getStride(),VERTEXBUFFER_STRIDE);
         fflush(stdin); fflush(stdout);
         return;
     }

    float *vertexData = (float*)vbuffer->devPtr;
    int nVertices = vbuffer->getVertexCount();
    dim3 cudaBlockSize(1024,1,1);
    dim3 cudaGridSize(nVertices/cudaBlockSize.x,1,1);

    extractGradientKernel<<<cudaGridSize,cudaBlockSize,0,vbuffer->cudaStream>>>(vertexData,vbuffer->getStride(),13,gradientScratchDev);
    checkCudaError("extractGradientMagnitudes error");
}
